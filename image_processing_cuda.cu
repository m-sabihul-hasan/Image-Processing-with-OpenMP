#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16  // CUDA block size

// Sobel filter kernels (Edge Detection)
__constant__ int Gx[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int Gy[3][3] = {
    {-1, -2, -1},
    {0, 0, 0},
    {1, 2, 1}
};

// Struct to hold image data
typedef struct {
    int width, height;
    unsigned char *data;
} Image;

// Function to read PPM file
void readPPM(const char *filename, Image *img) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) {
        printf("Error opening file!\n");
        exit(1);
    }

    char format[3];
    int maxColor;
    fscanf(fp, "%s\n%d %d\n%d\n", format, &img->width, &img->height, &maxColor);
    img->data = (unsigned char *)malloc(3 * img->width * img->height);
    fread(img->data, 3, img->width * img->height, fp);
    fclose(fp);
}

// Function to write PPM file
void writePPM(const char *filename, Image *img) {
    FILE *fp = fopen(filename, "wb");
    fprintf(fp, "P6\n%d %d\n255\n", img->width, img->height);
    fwrite(img->data, 3, img->width * img->height, fp);
    fclose(fp);
}

// CUDA Kernel for Blurring
__global__ void blurCUDA(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int sumR = 0, sumG = 0, sumB = 0;
        int count = 0;

        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                int idx = 3 * ((y + dy) * width + (x + dx));
                sumR += input[idx];
                sumG += input[idx + 1];
                sumB += input[idx + 2];
                count++;
            }
        }

        int idx = 3 * (y * width + x);
        output[idx] = sumR / count;
        output[idx + 1] = sumG / count;
        output[idx + 2] = sumB / count;
    }
}

// CUDA Kernel for Edge Detection (Sobel Filter)
__global__ void edgeDetectionCUDA(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int sumRx = 0, sumGx = 0, sumBx = 0;
        int sumRy = 0, sumGy = 0, sumBy = 0;

        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                int idx = 3 * ((y + dy) * width + (x + dx));
                int weightX = Gx[dy + 1][dx + 1];
                int weightY = Gy[dy + 1][dx + 1];

                sumRx += input[idx] * weightX;
                sumGx += input[idx + 1] * weightX;
                sumBx += input[idx + 2] * weightX;

                sumRy += input[idx] * weightY;
                sumGy += input[idx + 1] * weightY;
                sumBy += input[idx + 2] * weightY;
            }
        }

        int idx = 3 * (y * width + x);
        output[idx] = min(sqrtf(sumRx * sumRx + sumRy * sumRy), 255.0f);
        output[idx + 1] = min(sqrtf(sumGx * sumGx + sumGy * sumGy), 255.0f);
        output[idx + 2] = min(sqrtf(sumBx * sumBx + sumBy * sumBy), 255.0f);
    }
}

// Apply CUDA Image Processing with Timing
void applyCUDAFilter(Image *img, void (*kernel)(unsigned char*, unsigned char*, int, int)) {
    unsigned char *d_input, *d_output;
    size_t size = 3 * img->width * img->height * sizeof(unsigned char);

    // Allocate memory on GPU
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Copy data to GPU
    hipMemcpy(d_input, img->data, size, hipMemcpyHostToDevice);

    // Define grid and block sizes
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((img->width + BLOCK_SIZE - 1) / BLOCK_SIZE, (img->height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // CUDA timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Start timing
    hipEventRecord(start);

    // Launch CUDA kernel
    kernel<<<gridSize, blockSize>>>(d_input, d_output, img->width, img->height);
    hipDeviceSynchronize();

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Processing time: %.6f seconds\n", milliseconds / 1000.0);

    // Copy result back to CPU
    hipMemcpy(img->data, d_output, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_input);
    hipFree(d_output);
}

// Main Function
int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <input.ppm> <output.ppm> <filter>\n", argv[0]);
        printf("Filters: 1 (Blur), 2 (Edge Detection)\n");
        return 1;
    }

    Image img;
    readPPM(argv[1], &img);

    int filter_type = atoi(argv[3]);

    if (filter_type == 1) {
        applyCUDAFilter(&img, blurCUDA);
    } else if (filter_type == 2) {
        applyCUDAFilter(&img, edgeDetectionCUDA);
    } else {
        printf("Unknown filter type: %d\n", filter_type);
        free(img.data);
        return 1;
    }

    writePPM(argv[2], &img);
    free(img.data);
    return 0;
}